#include "hip/hip_runtime.h"
#include "rawspec.h"
#include "rawspec_version.h"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>

#define VERBOSE_ALLOC

#define NO_PLAN   ((hipfftHandle)-1)
#define NO_STREAM ((hipStream_t)-1)

#define LOAD_TEXTURE_WIDTH_POWER 15
#define LOAD_TEXTURE_WIDTH_MASK (unsigned int)((1<<LOAD_TEXTURE_WIDTH_POWER)-1)

#define MIN(a,b) ((a < b) ? (a) : (b))

#define PRINT_ERRMSG(error)                  \
  fprintf(stderr, "got error %s at %s:%d\n", \
      _cudaGetErrorEnum(error),  \
      __FILE__, __LINE__); fflush(stderr)

// Stream callback data structure
typedef struct {
  rawspec_context * ctx;
  int output_product;
} dump_cb_data_t;

// In full-Stokes mode (Npolout == -4) or full-pol mode (Npolout == 4), the
// CuFFT store callbacks are different depending on whether it is for pol0 or
// pol1:
//
// For full-Stokes mode, the store_callback_pol0_iquv function stores the
// voltage data into the first half of the 2x-sized FFT output buffer and
// accummulates (i.e. adds) the pol0 power into the first two quarters (I and
// Q) of the 4x-sized power buffer.
//
// For full-Stokes mode, the store_callback_pol1_iquv function accumulates
// (i.e. adds) the pol1 power into the first quarter of the 4x-sized power
// buffer (I), negatively accumulates (i.e. subtracts) the pol1 power into the
// second quarter of the 4x-sized power buffer (Q), reads the corresponding
// pol0 voltage from the first half of the 2x-sized FFT output buffer, and
// accumulates the complex pol0-pol1 power in the third (U) and fourth (V)
// quarters of the 4x-sized power buffer.
//
// For full-pol mode, the store_callback_pol0 function stores the voltage data
// into the first half of the 2x-sized FFT output buffer and accummulates the
// pol0 power into the first quarter of the 4x-sized power buffer.
//
// For full-pol mode, the store_callback_pol1 function accumulates the pol1
// power into the second quarter of the 4x-sized power buffer, reads the
// corresponding pol0 voltage from the first half of the 2x-sized FFT output
// buffer, accumulates the complex pol0-pol1 power in the third (real) and
// fourth (imaginary) quarters of the 4x-sized power buffer.
//
// We use a "store_cb_data_t" structure to pass device pointers to the
// various buffers involved.
typedef struct {
  hipfftComplex * fft_out_pol0;
  float * pwr_buf_p00_i;
  float * pwr_buf_p11_q;
  float * pwr_buf_p01_re_u;
  float * pwr_buf_p01_im_v;
} store_cb_data_t;

// GPU context structure
typedef struct {
  // Device pointer to FFT input buffer
  char * d_fft_in;
  // Device pointer to complex4 expansion LUT
  char2 * d_comp4_exp_LUT;
  // Device pointer to intermediary buffer for expansion of complex4 samples
  char * d_blk_expansion_buf;
  // Device pointer to FFT output buffer
  hipfftComplex * d_fft_out;
  // Array of device pointers to power buffers
  float * d_pwr_out[MAX_OUTPUTS];
  // Array of device pointers to incoherent-sum buffers
  float * d_ics_out[MAX_OUTPUTS];
  float * d_Aws;
  // Array of handles to FFT plans.
  // Each output product gets a pair of plans (one for each pol).
  hipfftHandle plan[MAX_OUTPUTS][2];
  // Array of device pointers to store_cb_data_t structures
  // (one per output product)
  store_cb_data_t *d_scb_data[MAX_OUTPUTS];
  // Device pointer to work area (shared by all plans!)
  void * d_work_area;
  // Size of work area
  size_t work_size;
  // Array of Ns values (number of specta (FFTs) per input buffer for Nt)
  unsigned int Nss[MAX_OUTPUTS];
  // Compute stream (as opposed to a "copy stream")
  hipStream_t compute_stream;
  // Array of grids for accumulate kernel
  dim3 grid[MAX_OUTPUTS];
  // Array of number of threads to use per block for accumulate kernel
  int nthreads[MAX_OUTPUTS];
  // Array of Ni values (number of input buffers per dump)
  unsigned int Nis[MAX_OUTPUTS];
  // A count of the number of input buffers processed
  unsigned int inbuf_count;
  // Array of dump_cb_data_t structures for dump callback
  dump_cb_data_t dump_cb_data[MAX_OUTPUTS];
  // CUDA Texture Object used to convert from integer to floating point
  hipTextureObject_t tex_obj;
  // CUDA Texture Object used to convert from complex4bit byte data to complex8bit short data
  hipTextureObject_t comp4_exp_tex_obj;
  // Flag indicating that the caller is managing the input block buffers
  // Non-zero when caller is managing (i.e. allocating and freeing) the
  // buffers; zero when we are.
  int caller_managed;
  // This is a commonly used value to stride between channels within GUPPI input-buffers,
  // the dimensionality of which is [channel (slowest), time, polarisation (fastest)]:
  // (ctx->Ntpb * ctx->Np * 2 /*complex*/ * ctx->Nbps)/8
  size_t guppi_channel_stride;
} rawspec_gpu_context;

// Device-side texture object declaration
__device__ hipTextureObject_t d_tex_obj;
__device__ hipTextureObject_t d_comp4_exp_tex_obj;

// The load_callback gets the input value through the texture memory to achieve
// a "for free" mapping of 8-bit integer values into 32-bit float values.
__device__ hipfftComplex load_callback(void *p_v_in,
                                      size_t offset,
                                      void *p_v_user,
                                      void *p_v_shared)
{
  hipfftComplex c;
  // p_v_in is input buffer (cast to hipfftComplex*) plus polarization offset.
  // p_v_user is input buffer.  offset is complex element offset from start of
  // input buffer, but does not include any polarization offset so we compute
  // the polarization offset by subtracting p_v_user from p_v_in and add it to
  // offset.
  offset += (hipfftComplex *)p_v_in - (hipfftComplex *)p_v_user;
  c.x = tex2D<float>(d_tex_obj, ((2*offset  ) & LOAD_TEXTURE_WIDTH_MASK), ((  offset  ) >> (LOAD_TEXTURE_WIDTH_POWER-1)));
  c.y = tex2D<float>(d_tex_obj, ((2*offset+1) & LOAD_TEXTURE_WIDTH_MASK), ((2*offset+1) >> LOAD_TEXTURE_WIDTH_POWER));
  return c;
}

// For total-power-only mode (Npolout == 1), the store_callback just needs to
// accumulate the power into the one and only power buffer.  It doesn't matter
// if it's for pol0 or pol1 since they all get added together eventually.
__device__ void store_callback(void *p_v_out,
                               size_t offset,
                               hipfftComplex element,
                               void *p_v_user,
                               void *p_v_shared)
{
  float pwr = element.x * element.x + element.y * element.y;
  ((float *)p_v_user)[offset] += pwr;
}

// For full-Stokes mode, the store_callback_pol0_iquv function stores the
// voltage data into the first half of the 2x-sized FFT output buffer and
// accummulates (i.e. adds) the pol0 power into the first two quarters (I and
// Q) of the 4x-sized power buffer.
__device__ void store_callback_pol0_iquv(void *p_v_out,
                                    size_t offset,
                                    hipfftComplex p0,
                                    void *p_v_user,
                                    void *p_v_shared)
{
  store_cb_data_t * d_scb_data = (store_cb_data_t *)p_v_user;
  float pwr = p0.x * p0.x + p0.y * p0.y;
  d_scb_data->pwr_buf_p00_i[offset] += pwr;
  d_scb_data->pwr_buf_p11_q[offset] += pwr;
  d_scb_data->fft_out_pol0[offset] = p0;
}

// For full-Stokes mode, the store_callback_pol1_iquv function accumulates
// (i.e. adds) the pol1 power into the first quarter of the 4x-sized power
// buffer (I), negatively accumulates (i.e. subtracts) the pol1 power into the
// second quarter of the 4x-sized power buffer (Q), reads the corresponding
// pol0 voltage from the first half of the 2x-sized FFT output buffer, and
// accumulates the complex pol0-pol1 power in the third (U) and fourth (V)
// quarters of the 4x-sized power buffer.
__device__ void store_callback_pol1_iquv(void *p_v_out,
                                    size_t offset,
                                    hipfftComplex p1,
                                    void *p_v_user,
                                    void *p_v_shared)
{
  store_cb_data_t * d_scb_data = (store_cb_data_t *)p_v_user;
  float pwr = p1.x * p1.x + p1.y * p1.y;
  d_scb_data->pwr_buf_p00_i[offset] += pwr;
  d_scb_data->pwr_buf_p11_q[offset] -= pwr;
  hipfftComplex p0 = d_scb_data->fft_out_pol0[offset];
  // TODO Verify sign and factor-of-two scaling for U and V
  d_scb_data->pwr_buf_p01_re_u[offset] += p0.x * p1.x + p0.y * p1.y;
  d_scb_data->pwr_buf_p01_im_v[offset] += p0.y * p1.x - p0.x * p1.y;
}

// Conjugated form of store_callback_pol1_iquv().
__device__ void store_callback_pol1_iquv_conj(void *p_v_out,
                                    size_t offset,
                                    hipfftComplex p1,
                                    void *p_v_user,
                                    void *p_v_shared)
{
  store_cb_data_t * d_scb_data = (store_cb_data_t *)p_v_user;
  float pwr = p1.x * p1.x + p1.y * p1.y;
  d_scb_data->pwr_buf_p00_i[offset] += pwr;
  d_scb_data->pwr_buf_p11_q[offset] -= pwr;
  hipfftComplex p0 = d_scb_data->fft_out_pol0[offset];
  // TODO Verify sign and factor-of-two scaling for U and V
  d_scb_data->pwr_buf_p01_re_u[offset] += p0.x * p1.x + p0.y * p1.y;
  d_scb_data->pwr_buf_p01_im_v[offset] -= p0.y * p1.x - p0.x * p1.y;
}

// For full-pol mode, the store_callback_pol0 function stores the voltage data
// into the first half of the 2x-sized FFT output buffer and accummulates the
// pol0 power into the first quarter of the 4x-sized power buffer.
__device__ void store_callback_pol0(void *p_v_out,
                                    size_t offset,
                                    hipfftComplex p0,
                                    void *p_v_user,
                                    void *p_v_shared)
{
  store_cb_data_t * d_scb_data = (store_cb_data_t *)p_v_user;
  float pwr = p0.x * p0.x + p0.y * p0.y;
  d_scb_data->pwr_buf_p00_i[offset] += pwr;
  d_scb_data->fft_out_pol0[offset] = p0;
}

// For full-pol mode, the store_callback_pol1 function accumulates the pol1
// power into the second quarter of the 4x-sized power buffer, reads the
// corresponding pol0 voltage from the first half of the 2x-sized FFT output
// buffer, accumulates the complex pol0-pol1 power in the third (real) and
// fourth (imaginary) quarters of the 4x-sized power buffer.
__device__ void store_callback_pol1(void *p_v_out,
                                    size_t offset,
                                    hipfftComplex p1,
                                    void *p_v_user,
                                    void *p_v_shared)
{
  store_cb_data_t * d_scb_data = (store_cb_data_t *)p_v_user;
  float pwr = p1.x * p1.x + p1.y * p1.y;
  d_scb_data->pwr_buf_p11_q[offset] += pwr;
  hipfftComplex p0 = d_scb_data->fft_out_pol0[offset];
  d_scb_data->pwr_buf_p01_re_u[offset] += p0.x * p1.x + p0.y * p1.y;
  d_scb_data->pwr_buf_p01_im_v[offset] += p0.y * p1.x - p0.x * p1.y;
}

// conjugated form of store_callback_pol1().
__device__ void store_callback_pol1_conj(void *p_v_out,
                                    size_t offset,
                                    hipfftComplex p1,
                                    void *p_v_user,
                                    void *p_v_shared)
{
  store_cb_data_t * d_scb_data = (store_cb_data_t *)p_v_user;
  float pwr = p1.x * p1.x + p1.y * p1.y;
  d_scb_data->pwr_buf_p11_q[offset] += pwr;
  hipfftComplex p0 = d_scb_data->fft_out_pol0[offset];
  d_scb_data->pwr_buf_p01_re_u[offset] += p0.x * p1.x + p0.y * p1.y;
  d_scb_data->pwr_buf_p01_im_v[offset] -= p0.y * p1.x - p0.x * p1.y;
}

__device__ hipfftCallbackLoadC d_cufft_load_callback = load_callback;
__device__ hipfftCallbackStoreC d_cufft_store_callback = store_callback;
__device__ hipfftCallbackStoreC d_cufft_store_callback_pol0 = store_callback_pol0;
__device__ hipfftCallbackStoreC d_cufft_store_callback_pol1 = store_callback_pol1;
__device__ hipfftCallbackStoreC d_cufft_store_callback_pol1_conj = store_callback_pol1_conj;
__device__ hipfftCallbackStoreC d_cufft_store_callback_pol0_iquv = store_callback_pol0_iquv;
__device__ hipfftCallbackStoreC d_cufft_store_callback_pol1_iquv = store_callback_pol1_iquv;
__device__ hipfftCallbackStoreC d_cufft_store_callback_pol1_iquv_conj = store_callback_pol1_iquv_conj;

#define MAX_THREADS (1024)

// Accumulate kernel
__global__ void accumulate(float * pwr_buf, unsigned int Na, size_t xpitch, size_t ypitch, size_t zpitch)
{
  unsigned int i;

  // TODO Add check for past end of spectrum

  off_t offset0 = blockIdx.z * zpitch
                + blockIdx.y * ypitch
                + blockIdx.x * MAX_THREADS
                + threadIdx.x;

  off_t offset = offset0;

  float sum = pwr_buf[offset];

  for(i=1; i<Na; i++) {
    offset += xpitch;
    sum += pwr_buf[offset];
  }

  pwr_buf[offset0] = sum;
}

// Incoherent summation kernel (across antenna)
__global__ void incoherent_sum(float * pwr_buf, float * incoh_buf, float * ant_weights, unsigned int Nant, size_t Nt,
                                size_t ant_pitch, size_t chan_pitch, size_t pol_pitch, size_t spectra_pitch,
                                size_t chan_out_pitch, size_t pol_out_pitch, size_t spectra_out_pitch
                              )
{
  const size_t coarse_chan_idx = (blockIdx.x* blockDim.x + threadIdx.x)/Nt;
  const size_t fine_chan_idx = (blockIdx.x* blockDim.x + threadIdx.x)%Nt;

  off_t offset_pwr =  blockIdx.z * spectra_pitch
                    + blockIdx.y * pol_pitch
                    + coarse_chan_idx * chan_pitch + fine_chan_idx;
  const off_t offset_ics =  blockIdx.z * spectra_out_pitch
                          + blockIdx.y * pol_out_pitch
                          + coarse_chan_idx * chan_out_pitch + fine_chan_idx + (fine_chan_idx < (Nt+1)/2 ? Nt/2 : -Nt/2);

  for(unsigned int i=0; i<Nant; i++) {
    incoh_buf[offset_ics] += ant_weights[i] * pwr_buf[offset_pwr];
    offset_pwr += ant_pitch;
  }
}

__global__ void complex4_expansion(char2 *lut){
  // The right shifts (>> 4) aren't perfectly necessary, as
  // with out them a scaling factor is introduced. They are kept
  // however, as the LUT only computes this 256 times, all in parallel,
  // and so no real speed gains are to be had.
  lut[blockIdx.x] = make_char2( ((char)(blockIdx.x&0xf0))>>4,       // Real component
                                ((char)((blockIdx.x&0x0f)<<4)) >> 4 // Imag component
                              );
}

// 4bit Expansion kernel
// Takes the half full blocks of the gpu_ctx->d_blk_expansion_buf buffer and expands 
// each complex4 byte. The transferal mimics the hipMemcpy2D of 
// rawspec_copy_blocks_to_gpu:
// The src order is [time, channel, blocks]  (fastest --> slowest)
// The dst order is [time, blocks, channels] (fastest --> slowest)
//
// Expectation of blockDim, with ctx->Np threads each:
// grid.x = ctx->Ntpb;
// grid.y = ctx->Nc;
// grid.z = num_blocks;
__global__ void copy_expand_complex4(char *comp8_dst, char *comp4_src, size_t num_blocks,
                                     size_t block_pitch, size_t channel_pitch)
{                                     
  char* comp8_dst_offset = comp8_dst + 2*(blockIdx.y*num_blocks*channel_pitch +
                                          blockIdx.z*channel_pitch +
                                          blockIdx.x*blockDim.x + threadIdx.x);
  const char2 comp8 = tex1Dfetch<char2>(d_comp4_exp_tex_obj, (unsigned char) (comp4_src[blockIdx.z*block_pitch + 
                                                                      blockIdx.y*channel_pitch +
                                                                      blockIdx.x*blockDim.x + threadIdx.x]));
  comp8_dst_offset[0] = comp8.x;
  comp8_dst_offset[1] = comp8.y;
}

// Stream callback function that is called right before an output product's GPU
// power buffer has been copied to the host power buffer.
static void CUDART_CB pre_dump_stream_callback(hipStream_t stream,
                                               hipError_t status,
                                               void *data)
{
  dump_cb_data_t * dump_cb_data = (dump_cb_data_t *)data;
  if(dump_cb_data->ctx->dump_callback) {
    dump_cb_data->ctx->dump_callback(dump_cb_data->ctx,
                                     dump_cb_data->output_product,
                                     RAWSPEC_CALLBACK_PRE_DUMP);
  }
}

// Stream callback function that is called right after an output product's GPU
// power buffer has been copied to the host power buffer.
static void CUDART_CB post_dump_stream_callback(hipStream_t stream,
                                                hipError_t status,
                                                void *data)
{
  dump_cb_data_t * dump_cb_data = (dump_cb_data_t *)data;
  if(dump_cb_data->ctx->dump_callback) {
    dump_cb_data->ctx->dump_callback(dump_cb_data->ctx,
                                     dump_cb_data->output_product,
                                     RAWSPEC_CALLBACK_POST_DUMP);
  }
}

// This stringification trick is from "info cpp"
#define STRINGIFY1(s) #s
#define STRINGIFY(s) STRINGIFY1(s)
static const char rawspec_version[] = STRINGIFY(RAWSPEC_VERSION) " cuFFT"
#ifdef CUFFT_VER_MAJOR
  " " STRINGIFY(CUFFT_VER_MAJOR)
#ifdef CUFFT_VER_MINOR
  "." STRINGIFY(CUFFT_VER_MINOR)
#ifdef CUFFT_VER_PATCH
  "." STRINGIFY(CUFFT_VER_PATCH)
#ifdef CUFFT_VER_BUILD
  "." STRINGIFY(CUFFT_VER_BUILD)
#endif // CUFFT_VER_BUILD
#endif // CUFFT_VER_PATCH
#endif // CUFFT_VER_MINOR
#else
  " unknown/old"
#endif // CUFFT_VER_MAJOR
;

// Returns a pointer to a string containing the rawspec version
const char * rawspec_version_string()
{
  return rawspec_version;
}

// Sets ctx->Ntmax.
// Allocates host and device buffers based on the ctx->N values.
// Allocates and sets the ctx->gpu_ctx field.
// Creates CuFFT plans.
// Creates streams.
// Returns 0 on success, non-zero on error.
int rawspec_initialize(rawspec_context * ctx)
{
  int i;
  int p;
  // A simple bool-flag for now, but could rather hold expansion ratio
  // ^^ would require appropriate changes in rawspec.c (see expand4bps_to8bps)
  char NbpsIsExpanded = 0;
  uint64_t buf_size;
  size_t work_size = 0;
  store_cb_data_t h_scb_data;
  hipError_t cuda_rc;
  hipfftResult cufft_rc;
  hipResourceDesc res_desc;
  hipTextureDesc tex_desc;
  int texture_attribute_maximum;


  // Host copies of cufft callback pointers
  hipfftCallbackLoadC h_cufft_load_callback;
  hipfftCallbackStoreC h_cufft_store_callback;
  hipfftCallbackStoreC h_cufft_store_callback_pols[2];
  hipfftCallbackStoreC h_cufft_store_callback_iquv[2];

  // Validate No
  if(ctx->No == 0 || ctx->No > MAX_OUTPUTS) {
    fprintf(stderr, "output products must be in range [1..%d], not %d\n",
        MAX_OUTPUTS, ctx->No);
    fflush(stderr);
    return 1;
  }

  // Validate Np
  if(ctx->Np == 0 || ctx->Np > 2) {
    fprintf(stderr,
        "number of polarizations must be in range [1..2], not %d\n", ctx->Np);
    fflush(stderr);
    return 1;
  }

  // Validate/set Npolout values
  for(i=0; i<ctx->No; i++) {
    if(abs(ctx->Npolout[i]) != 4 || ctx->Np != 2) {
      ctx->Npolout[i] = 1;
    }
  }

  // Validate Ntpb
  if(ctx->Ntpb == 0) {
    fprintf(stderr, "number of time samples per block cannot be zero\n");
    fflush(stderr);
    return 1;
  }

  // Validate Nbps. Zero silently defaults to 8 for backwards compatibility
  // with pre-Nbps versions.  Any other value except 8 or 16 is treated as 8
  // and a warning is issued to stderr.
  if(ctx->Nbps == 0) {
    ctx->Nbps = 8;
  } else if(ctx->Nbps != 8 && ctx->Nbps != 16) {
    fprintf(stderr,
        "number of bits per sample must be 8 or 16 (not %d), using 8 bps\n",
        ctx->Nbps);
    fflush(stderr);
    NbpsIsExpanded = ctx->Nbps == 4;
    ctx->Nbps = 8;
  }

  // Determine Ntmax (and validate Nts)
  ctx->Ntmax = 0;
  for(i=0; i<ctx->No; i++) {
    if(ctx->Nts[i] == 0) {
      fprintf(stderr, "Nts[%d] cannot be 0\n", i);
      fflush(stderr);
      return 1;
    }
    if(ctx->Ntmax < ctx->Nts[i]) {
      ctx->Ntmax = ctx->Nts[i];
    }
  }
  // Validate that all Nts are factors of Ntmax.  This constraint helps
  // simplify input buffer management.
  for(i=0; i<ctx->No; i++) {
    if(ctx->Ntmax % ctx->Nts[i] != 0) {
      fprintf(stderr, "Nts[%d] (%u) is not a factor of Ntmax (%u)\n",
          i, ctx->Nts[i], ctx->Ntmax);
      fflush(stderr);
      return 1;
    }
  }

  // Validate/calculate Nb
  // If ctx->Nb is given by caller (i.e. is non-zero)
  if(ctx->Nb != 0) {
    // Validate that Ntmax is a factor of (Nb * Ntpb)
    if((ctx->Nb * ctx->Ntpb) % ctx->Ntmax != 0) {
      fprintf(stderr,
          "Ntmax (%u) is not a factor of Nb*Ntpb (%u * %u = %u)\n",
          ctx->Ntmax, ctx->Nb, ctx->Ntpb, ctx->Nb*ctx->Ntpb);
      fflush(stderr);
      return 1;
    }
  } else {
    // Calculate Nb
    // If Ntmax is less than one block
    if(ctx->Ntmax < ctx->Ntpb) {
      // Validate that Ntmax is a factor of Ntpb
      if(ctx->Ntpb % ctx->Ntmax != 0) {
        fprintf(stderr, "Ntmax (%u) is not a factor of Ntpb (%u)\n",
            ctx->Ntmax, ctx->Ntpb);
        fflush(stderr);
        return 1;
      }
      ctx->Nb = 1;
    } else {
      // Validate that Ntpb is factor of Ntmax
      if(ctx->Ntmax % ctx->Ntpb != 0) {
        fprintf(stderr, "Ntpb (%u) is not a factor of Nmax (%u)\n",
            ctx->Ntpb, ctx->Ntmax);
        fflush(stderr);
        return 1;
      }
      ctx->Nb = ctx->Ntmax / ctx->Ntpb;
    }
  }

  // Ensure Nb_host is non-zero when host input buffers are caller managed
  if(ctx->Nb_host == 0 && ctx->h_blkbufs) {
    fprintf(stderr,
        "Must specify number of host input blocks when caller-managed\n");
    fflush(stderr);
    return 1;
  } else if(ctx->Nb_host == 0) {
    ctx->Nb_host = ctx->Nb;
  }

  // Validate Nas
  for(i=0; i < ctx->No; i++) {
    if(ctx->Nas[i] == 0) {
      fprintf(stderr, "Nas[%d] cannot be 0\n", i);
      fflush(stderr);
      return 1;
    }
    // If mulitple integrations per input buffer
    if(ctx->Nts[i]*ctx->Nas[i] < ctx->Nb*ctx->Ntpb) {
      // Must have integer integrations per input buffer
      if((ctx->Nb * ctx->Ntpb) % (ctx->Nts[i] * ctx->Nas[i]) != 0) {
        fprintf(stderr,
            "Nts[%d] * Nas[%d] (%u * %u) must divide Nb * Ntpb (%u * %u)\n",
            i, i, ctx->Nts[i], ctx->Nas[i], ctx->Nb, ctx->Ntpb);
        fflush(stderr);
        return 1;
      }
    } else {
      // Must have integer input buffers per integration
      if((ctx->Nts[i] * ctx->Nas[i]) % (ctx->Nb * ctx->Ntpb) != 0) {
        fprintf(stderr,
            "Nb * Ntpb (%u * %u) must divide Nts[%d] * Nas[%d] (%u * %u)\n",
            ctx->Nb, ctx->Ntpb, i, i, ctx->Nts[i], ctx->Nas[i]);
        fflush(stderr);
        return 1;
      }
    }
  }

  ctx->Nant = ctx->Nant <= 0 ? 1 : ctx->Nant;
  // Setup channel-chunk parametners
  if(ctx->Ncc == 0) { // Disable channel-chunking
    ctx->Ncc = ctx->Nc;
  }
  else{ // Enabled channel-chunking
    if(ctx->Ncc <= 1) { // Auto channel-chunking
      if(ctx->Nant > 1){
        ctx->Ncc = ctx->Nc/ctx->Nant;
      }
      else{ // find largest Nc factor <= 10
        for(i = 1; i <= 10; i++){
          if(ctx->Nc%i == 0){
            ctx->Ncc = ctx->Nc/i;
          }
        }
      }
    }
    else if(ctx->Nc%ctx->Ncc != 0) { // Manual channel-chunking, but inappropriate chunks
      fprintf(stderr, "%d channels cannot be factorised to chunks of %d\n",
        ctx->Nc, ctx->Ncc
      );
      return 1;
    }

    printf("Chunking %d channels into %d chunks of %d.\n", ctx->Nc, ctx->Nc/ctx->Ncc, ctx->Ncc);
  }

  // Null out all pointers
  // TODO Add support for client managed host buffers
  for(i=0; i < MAX_OUTPUTS; i++) {
    ctx->h_pwrbuf[i] = NULL;
    ctx->h_icsbuf[i] = NULL;
  }
  ctx->gpu_ctx = NULL;

  // Set CUDA device (validates gpu_index)
  cuda_rc = hipSetDevice(ctx->gpu_index);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    // TODO return distinct error code
    return 1;
  }

  // Allocate GPU context
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)malloc(sizeof(rawspec_gpu_context));

  if(!gpu_ctx) {
    fprintf(stderr, "unable to allocate %lu bytes for rawspec GPU context\n",
        sizeof(rawspec_gpu_context));
    fflush(stderr);
    rawspec_cleanup(ctx);
    return 1;
  }

  // Store pointer to gpu_ctx in ctx
  ctx->gpu_ctx = gpu_ctx;

  // NULL out pointers (and invalidate plans)
  gpu_ctx->d_fft_in = NULL;
  gpu_ctx->d_comp4_exp_LUT = NULL;
  gpu_ctx->d_blk_expansion_buf = NULL;
  gpu_ctx->d_fft_out = NULL;
  gpu_ctx->d_work_area = NULL;
  gpu_ctx->work_size = 0;
  gpu_ctx->compute_stream = NO_STREAM;
  for(i=0; i<MAX_OUTPUTS; i++) {
    gpu_ctx->d_pwr_out[i] = NULL;
    gpu_ctx->d_scb_data[i] = NULL;
    gpu_ctx->plan[i][0] = NO_PLAN;
    gpu_ctx->plan[i][1] = NO_PLAN;
    gpu_ctx->dump_cb_data[i].ctx = ctx;
    gpu_ctx->dump_cb_data[i].output_product = i;
  }

  // Initialize inbuf_count
  gpu_ctx->inbuf_count = 0;

  if(!ctx->h_blkbufs) {
    // Remember that we (not the caller) are managing these buffers
    // (i.e. we will need to free them when cleaning up).
    gpu_ctx->caller_managed = 0;

    // Alllocate host input block buffers
    ctx->h_blkbufs = (char **)malloc(ctx->Nb_host * sizeof(char *));
    for(i=0; i < ctx->Nb_host; i++) {
      // Block buffer can use write combining
      cuda_rc = hipHostAlloc(&ctx->h_blkbufs[i],
          ctx->Ntpb * ctx->Np * ctx->Nc * 2 /*complex*/ * (ctx->Nbps/8),
          hipHostMallocWriteCombined);
      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        rawspec_cleanup(ctx);
        return 1;
      }
    }
  } else {
    // Remember that the caller is managing these buffers
    // (i.e. we will only need to unregister them when cleaning up).
    gpu_ctx->caller_managed = 1;

    // Register these buffers with CUDA.  It is the caller's responsibility to
    // ensure that the blocks meet memory alignment requirements, etc.
    for(i=0; i < ctx->Nb_host; i++) {
      cuda_rc = hipHostRegister(ctx->h_blkbufs[i],
          ctx->Ntpb * ctx->Np * ctx->Nc * 2 /*complex*/ * (ctx->Nbps/8),
          hipHostRegisterDefault);
      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        rawspec_cleanup(ctx);
        return 1;
      }
    }
  }

  // Calculate Ns and allocate host power output buffers
  for(i=0; i < ctx->No; i++) {
    // Ns[i] is number of specta (FFTs) per coarse channel for one input buffer
    // for Nt[i] points per spectra.
    gpu_ctx->Nss[i] = (ctx->Nb * ctx->Ntpb) / ctx->Nts[i];

    // Calculate number of spectra per dump
    ctx->Nds[i] = gpu_ctx->Nss[i] / ctx->Nas[i];
    if(ctx->Nds[i] == 0) {
      ctx->Nds[i] = 1;
    }

    // Calculate number of input buffers per dump
    gpu_ctx->Nis[i] = ctx->Nas[i] / gpu_ctx->Nss[i];
    if(gpu_ctx->Nis[i] == 0) {
      gpu_ctx->Nis[i] = 1;
    }

    // Calculate grid dimensions
    gpu_ctx->grid[i].x = (ctx->Nts[i] + MAX_THREADS - 1) / MAX_THREADS;
    gpu_ctx->grid[i].y = ctx->Nds[i];
    gpu_ctx->grid[i].z = ctx->Ncc;

    // Calculate number of threads per block
    gpu_ctx->nthreads[i] = MIN(ctx->Nts[i], MAX_THREADS);

    // Host buffer needs to accommodate the number of integrations that will be
    // dumped at one time (Nd).
    ctx->h_pwrbuf_size[i] = abs(ctx->Npolout[i]) *
                            ctx->Nds[i]*ctx->Nts[i]*ctx->Nc*sizeof(float);
    #ifdef VERBOSE_ALLOC
      printf("FFT Host dump buffer[%d] size == %lu\n", i, ctx->h_pwrbuf_size[i]);
    #endif
    cuda_rc = hipHostAlloc(&ctx->h_pwrbuf[i], ctx->h_pwrbuf_size[i],
                       hipHostMallocDefault);

    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
    if(ctx->incoherently_sum == 1){// TODO validate that Nant > 1
      cuda_rc = hipHostAlloc(&ctx->h_icsbuf[i], ctx->h_pwrbuf_size[i]/ctx->Nant,
                        hipHostMallocDefault);

      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        rawspec_cleanup(ctx);
        return 1;
      }
    }
  }

  gpu_ctx->guppi_channel_stride = (ctx->Ntpb * ctx->Np * 2 /*complex*/ * ctx->Nbps)/8;

  // Allocate buffers

  // FFT input buffer
  // The input buffer is padded to the next multiple of 1<<LOAD_TEXTURE_WIDTH_POWER
  // to facilitate 2D texture lookups by treating the input buffer as a 2D array
  // that is 1<<LOAD_TEXTURE_WIDTH_POWER wide.
  buf_size = ctx->Nb*ctx->Nc*gpu_ctx->guppi_channel_stride;
  if((buf_size & LOAD_TEXTURE_WIDTH_MASK) != 0) {
    // Round up to next multiple of 64KB
    buf_size = (buf_size & ~LOAD_TEXTURE_WIDTH_MASK) + 1<<LOAD_TEXTURE_WIDTH_POWER;
  }

#ifdef VERBOSE_ALLOC
  printf("FFT input buffer size == %lu\n", buf_size);
#endif
  cuda_rc = hipMalloc(&gpu_ctx->d_fft_in, buf_size);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  
  hipDeviceGetAttribute(&texture_attribute_maximum, hipDeviceAttributeMaxTexture2DLinear, ctx->gpu_index);
  if(texture_attribute_maximum < 1<<LOAD_TEXTURE_WIDTH_POWER){
    fprintf(stderr, "Maximum 2D texture width: %d.\n", texture_attribute_maximum);
    fprintf(stderr, "\tThe static load-texture-width of 1<<LOAD_TEXTURE_WIDTH_POWER exceeds this: %d\n", 1<<LOAD_TEXTURE_WIDTH_POWER);
    fprintf(stderr, "\tExpect a CUDA raised failure!\n");
  }
  hipDeviceGetAttribute(&texture_attribute_maximum, hipDeviceAttributeMaxTexture2DLinear, ctx->gpu_index);
  if(texture_attribute_maximum < buf_size>>LOAD_TEXTURE_WIDTH_POWER){
    fprintf(stderr, "Maximum 2D texture height: %d.\n", texture_attribute_maximum);
    fprintf(stderr, "\tThe load-texture-height of `buf_size (%lu)>>(%d) LOAD_TEXTURE_WIDTH_POWER` exceeds this: %lu\n", buf_size, LOAD_TEXTURE_WIDTH_POWER, buf_size>>LOAD_TEXTURE_WIDTH_POWER);
    fprintf(stderr, "\tExpect a CUDA raised failure!\n");

    hipDeviceGetAttribute(&texture_attribute_maximum, hipDeviceAttributeMaxTexture2DLinear, ctx->gpu_index);
    if(texture_attribute_maximum > 1<<LOAD_TEXTURE_WIDTH_POWER){
      fprintf(stderr, "\tLOAD_TEXTURE_WIDTH_POWER could be increased to %d (at most) to possibly circumvent this issued\n", 31 - __builtin_clz(texture_attribute_maximum));
    }
  }
  hipDeviceGetAttribute(&texture_attribute_maximum, hipDeviceAttributeMaxTexture2DLinear, ctx->gpu_index);
  if(texture_attribute_maximum < (1<<LOAD_TEXTURE_WIDTH_POWER) * (ctx->Nbps/8)){
    fprintf(stderr, "Maximum 2D texture pitch: %d.\n", texture_attribute_maximum);
    fprintf(stderr, "\tThe load-texture-pitch of (1<<LOAD_TEXTURE_WIDTH_POWER) * (ctx->Nbps/8) exceeds this: %d\n", (1<<LOAD_TEXTURE_WIDTH_POWER) * (ctx->Nbps/8));
    fprintf(stderr, "\tExpect a CUDA raised failure!\n");
  }
  fflush(stderr);

  // Create texture object for device input buffer
  // res_desc describes input resource
  // Width is 32K elements, height is buf_size/32K elements, pitch is 32K elements
  memset(&res_desc, 0, sizeof(res_desc));
  res_desc.resType = hipResourceTypePitch2D;
  res_desc.res.pitch2D.devPtr = gpu_ctx->d_fft_in;
  res_desc.res.pitch2D.desc.f = hipChannelFormatKindSigned;
  res_desc.res.pitch2D.desc.x = ctx->Nbps; // bits per sample
  res_desc.res.pitch2D.width = 1<<LOAD_TEXTURE_WIDTH_POWER;         // elements
  res_desc.res.pitch2D.height = buf_size>>LOAD_TEXTURE_WIDTH_POWER; // elements
  res_desc.res.pitch2D.pitchInBytes = (1<<LOAD_TEXTURE_WIDTH_POWER) * (ctx->Nbps/8);  // bytes!
  // tex_desc describes texture mapping
  memset(&tex_desc, 0, sizeof(tex_desc));
#if 0 // These settings are not used in online examples involved hipReadModeNormalizedFloat
  // Not sure whether address_mode matters for hipReadModeNormalizedFloat
  tex_desc.address_mode[0] = hipAddressModeClamp;
  tex_desc.address_mode[1] = hipAddressModeClamp;
  tex_desc.address_mode[2] = hipAddressModeClamp;
  // Not sure whether filter_mode matters for hipReadModeNormalizedFloat
  tex_desc.filter_mode = hipFilterModePoint;
#endif // 0
  tex_desc.readMode = hipReadModeNormalizedFloat;

  cuda_rc = hipCreateTextureObject(&gpu_ctx->tex_obj,
                                    &res_desc, &tex_desc, NULL);

  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  // Copy texture object to device
  cuda_rc = hipMemcpyToSymbol(HIP_SYMBOL(d_tex_obj),
                               &gpu_ctx->tex_obj,
                               sizeof(hipTextureObject_t));

  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  if(NbpsIsExpanded){
#ifdef VERBOSE_ALLOC
    printf("NBITS expansion buffer size == %lu\n", buf_size/2);
#endif
    cuda_rc = hipMalloc(&gpu_ctx->d_blk_expansion_buf, buf_size/2);
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }

#ifdef VERBOSE_ALLOC
    printf("Complex4 expansion LUT size == %lu\n", 256*sizeof(char2));
#endif
    cuda_rc = hipMalloc(&gpu_ctx->d_comp4_exp_LUT, 256*sizeof(char2));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
    complex4_expansion<<<256,1>>>(gpu_ctx->d_comp4_exp_LUT);

    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeLinear;
    res_desc.res.linear.devPtr = gpu_ctx->d_comp4_exp_LUT;
    res_desc.res.linear.desc.f = hipChannelFormatKindSigned;
    res_desc.res.linear.desc.x = 8; // bits per channel
    res_desc.res.linear.desc.y = 8; // bits per channel
    res_desc.res.linear.sizeInBytes = 256*sizeof(char2);

    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.readMode = hipReadModeElementType;
  
    cuda_rc = hipCreateTextureObject(&gpu_ctx->comp4_exp_tex_obj,
                                      &res_desc, &tex_desc, NULL);
  
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
  
    cuda_rc = hipMemcpyToSymbol(HIP_SYMBOL(d_comp4_exp_tex_obj),
                                &gpu_ctx->comp4_exp_tex_obj,
                                sizeof(hipTextureObject_t));

    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
  }

  // FFT output buffer
  buf_size = ctx->Nb*ctx->Ntpb*ctx->Ncc*sizeof(hipfftComplex);
  // If any output product is full-pol then we need to double output buffer
  for(i=0; i < ctx->No; i++) {
    if(abs(ctx->Npolout[i]) == 4) {
      buf_size *= 2;
      break;
    }
  }
#ifdef VERBOSE_ALLOC
  printf("FFT output buffer size == %lu\n", buf_size);
#endif
  cuda_rc = hipMalloc(&gpu_ctx->d_fft_out, buf_size);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  // For each output product
  for(i=0; i < ctx->No; i++) {
    // Power output buffer
#ifdef VERBOSE_ALLOC
    printf("Power output buffer size == %u * %lu == %lu\n",
        abs(ctx->Npolout[i]),  ctx->Nb*ctx->Ntpb*ctx->Ncc*sizeof(float),
        abs(ctx->Npolout[i]) * ctx->Nb*ctx->Ntpb*ctx->Ncc*sizeof(float));
#endif
    cuda_rc = hipMalloc(&gpu_ctx->d_pwr_out[i],
        abs(ctx->Npolout[i]) * ctx->Nb*ctx->Ntpb*ctx->Ncc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
    // Clear power output buffer
    cuda_rc = hipMemset(gpu_ctx->d_pwr_out[i], 0,
        abs(ctx->Npolout[i]) * ctx->Nb*ctx->Ntpb*ctx->Ncc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }

    if(ctx->incoherently_sum){
#ifdef VERBOSE_ALLOC
      printf("ICS output buffer size == %u * %lu / %u == %lu\n",
          abs(ctx->Npolout[i]),  ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float), ctx->Nant,
          abs(ctx->Npolout[i]) * ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float)/ctx->Nant);
#endif
      cuda_rc = hipMalloc(&gpu_ctx->d_ics_out[i],
          abs(ctx->Npolout[i]) * ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float)/ctx->Nant);
      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        rawspec_cleanup(ctx);
        return 1;
      }
      // Clear incoherent-sum output buffer
      cuda_rc = hipMemset(gpu_ctx->d_ics_out[i], 0,
          abs(ctx->Npolout[i]) * ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float)/ctx->Nant);
      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        rawspec_cleanup(ctx);
        return 1;
      }

      // Setup device antenna-weight buffer
#ifdef VERBOSE_ALLOC
      printf("ICS antenna-weight buffer size == %lu\n", ctx->Nant*sizeof(float));
#endif
      cuda_rc = hipMalloc(&gpu_ctx->d_Aws, ctx->Nant*sizeof(float));
      if(cuda_rc != hipSuccess) {
        PRINT_ERRMSG(cuda_rc);
        rawspec_cleanup(ctx);
        return 1;
      }

      if(ctx->Naws == 1 && ctx->Naws < ctx->Nant){
        printf("Using the single antenna-weight (%f) for all antennas in the incoherent-sum.\n", ctx->Aws[0]);
        for(int w = 0; w < ctx->Nant; w++){
          hipMemcpy(gpu_ctx->d_Aws+w, ctx->Aws, sizeof(float), hipMemcpyHostToDevice);
        }
      }
      else if(ctx->Naws == ctx->Nant){
        for(int w = 0; w < ctx->Nant; w++){
          hipMemcpy(gpu_ctx->d_Aws+w, ctx->Aws + w, sizeof(float), hipMemcpyHostToDevice);
        }
      }
      else{
        fprintf(stderr, "Not enough antenna-weights provided for the %d antennas: only provided %d.\n", ctx->Nant, ctx->Naws);
        rawspec_cleanup(ctx);
        return 1;
      }
    }
    // Save pointer to FFT output buffer in store_cb_data
    h_scb_data.fft_out_pol0 = gpu_ctx->d_fft_out;
    // Save pointers into power ouput buffer
    h_scb_data.pwr_buf_p00_i = gpu_ctx->d_pwr_out[i];
    // These next fields are only used if abs(Npolout) == 4,
    // so we can initialize them that way even if Npolout == 1
    // (because they will never be used). It might be slightly
    // safer to init them to the same as pwr_buf_p00_i if Npolout == 1.
    h_scb_data.pwr_buf_p11_q =
        gpu_ctx->d_pwr_out[i] + 1*ctx->Nb*ctx->Ntpb*ctx->Ncc;
    h_scb_data.pwr_buf_p01_re_u =
        gpu_ctx->d_pwr_out[i] + 2*ctx->Nb*ctx->Ntpb*ctx->Ncc;
    h_scb_data.pwr_buf_p01_im_v =
        gpu_ctx->d_pwr_out[i] + 3*ctx->Nb*ctx->Ntpb*ctx->Ncc;

    // Allocate device memory for store_cb_data_t array
    cuda_rc = hipMalloc(&gpu_ctx->d_scb_data[i], sizeof(store_cb_data_t));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }

    // Copy store_cb_data_t arary from host to device
    cuda_rc = hipMemcpy(gpu_ctx->d_scb_data[i],
                         &h_scb_data,
                         sizeof(store_cb_data_t),
                         hipMemcpyHostToDevice);
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      rawspec_cleanup(ctx);
      return 1;
    }
  }

  // Get host pointers to cufft callbacks
  cuda_rc = hipMemcpyFromSymbol(&h_cufft_load_callback,
                                 HIP_SYMBOL(d_cufft_load_callback),
                                 sizeof(h_cufft_load_callback));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  cuda_rc = hipMemcpyFromSymbol(&h_cufft_store_callback,
                                 HIP_SYMBOL(d_cufft_store_callback),
                                 sizeof(h_cufft_store_callback));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  cuda_rc = hipMemcpyFromSymbol(&h_cufft_store_callback_pols[0],
                                 HIP_SYMBOL(d_cufft_store_callback_pol0),
                                 sizeof(h_cufft_store_callback_pols[0]));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  cuda_rc = hipMemcpyFromSymbol(&h_cufft_store_callback_pols[1],
                                 HIP_SYMBOL(ctx->input_conjugated ? d_cufft_store_callback_pol1_conj
                                                       : d_cufft_store_callback_pol1),
                                 sizeof(h_cufft_store_callback_pols[1]));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  cuda_rc = hipMemcpyFromSymbol(&h_cufft_store_callback_iquv[0],
                                 HIP_SYMBOL(d_cufft_store_callback_pol0_iquv),
                                 sizeof(h_cufft_store_callback_iquv[0]));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  cuda_rc = hipMemcpyFromSymbol(&h_cufft_store_callback_iquv[1],
                                 HIP_SYMBOL(ctx->input_conjugated ? d_cufft_store_callback_pol1_iquv_conj
                                                       : d_cufft_store_callback_pol1_iquv),
                                 sizeof(h_cufft_store_callback_iquv[1]));
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  // Create the "compute stream"
  cuda_rc = hipStreamCreateWithFlags(&gpu_ctx->compute_stream,
                                      hipStreamNonBlocking);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }

  // Generate FFT plans and associate callbacks and stream
  for(i=0; i < ctx->No; i++) {
    for(p=0; p<2; p++) {
      // Create plan handle (does not "make the plan", that happens later)
      cufft_rc = hipfftCreate(&gpu_ctx->plan[i][p]);
      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }

      // Prevent auto-allocation of work area for plan
      cufft_rc = hipfftSetAutoAllocation(gpu_ctx->plan[i][p], 0);
      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }

#ifdef VERBOSE_ALLOC
      printf("hipfftMakePlanMany for output product %d...", i);
#endif
      // Make the plan
      // TODO Are sizes here in units of elements or bytes?  Assume elements for now...
      cufft_rc = hipfftMakePlanMany(
                      gpu_ctx->plan[i][p],     // plan handle
                      1,                       // rank
                      (int *)&ctx->Nts[i],     // *n
                      (int *)&ctx->Nts[i],     // *inembed (unused for 1d)
                      ctx->Np,                 // istride
                      ctx->Nts[i]*ctx->Np,     // idist
                      (int *)&ctx->Nts[i],     // *onembed (unused for 1d)
                      1,                       // ostride
                      ctx->Nts[i],             // odist
                      HIPFFT_C2C,               // type
                      gpu_ctx->Nss[i]*ctx->Ncc,// batch
                      &work_size               // work area size
                 );

      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }
#ifdef VERBOSE_ALLOC
      printf("ok\n");
#endif

      // Now associate the callbacks with the plan.
      // Load callback
      cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i][p],
                                    (void **)&h_cufft_load_callback,
                                    HIPFFT_CB_LD_COMPLEX,
                                    (void **)&gpu_ctx->d_fft_in);
      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }
      // Store callback(s)
      if(ctx->Npolout[i] == 1) {
        cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i][p],
                                      (void **)&h_cufft_store_callback,
                                      HIPFFT_CB_ST_COMPLEX,
                                      (void **)&gpu_ctx->d_pwr_out[i]);
      } else if(ctx->Npolout[i] == 4) {
        cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i][p],
                                      (void **)&h_cufft_store_callback_pols[p],
                                      HIPFFT_CB_ST_COMPLEX,
                                      (void **)&gpu_ctx->d_scb_data[i]);
      } else if(ctx->Npolout[i] == -4) {
        cufft_rc = hipfftXtSetCallback(gpu_ctx->plan[i][p],
                                      (void **)&h_cufft_store_callback_iquv[p],
                                      HIPFFT_CB_ST_COMPLEX,
                                      (void **)&gpu_ctx->d_scb_data[i]);
      } else {
        fprintf(stderr, "invalid Npolout[%d]: %d\n", i, ctx->Npolout[i]);
        fflush(stderr);
        return 1;
      }
      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }

      // Associate compute stream with plan
      cufft_rc = hipfftSetStream(gpu_ctx->plan[i][p], gpu_ctx->compute_stream);
      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }

      // Get work size for this plan
      cufft_rc = hipfftGetSize(gpu_ctx->plan[i][p], &work_size);
      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }

      // Save size if it's largest one so far
      if(gpu_ctx->work_size < work_size) {
        gpu_ctx->work_size = work_size;
      }
    }
  }

  // Allocate work area
#ifdef VERBOSE_ALLOC
  printf("allocating work area %lu bytes...", work_size);
#endif
  cuda_rc = hipMalloc(&gpu_ctx->d_work_area, gpu_ctx->work_size);
  if(cuda_rc != hipSuccess) {
    PRINT_ERRMSG(cuda_rc);
    rawspec_cleanup(ctx);
    return 1;
  }
#ifdef VERBOSE_ALLOC
  printf("ok\n");
#endif

  // Associate work area with plans
  for(i=0; i < ctx->No; i++) {
    for(p=0; p<2; p++) {
      cufft_rc = hipfftSetWorkArea(gpu_ctx->plan[i][p], gpu_ctx->d_work_area);
      if(cufft_rc != HIPFFT_SUCCESS) {
        PRINT_ERRMSG(cufft_rc);
        rawspec_cleanup(ctx);
        return 1;
      }
    }
  }

  return 0;
}

// Frees host and device buffers based on the ctx->N values.
// Frees and sets the ctx->rawspec_gpu_ctx field.
// Destroys CuFFT plans.
// Destroys streams.
void rawspec_cleanup(rawspec_context * ctx)
{
  int i;
  int p;
  rawspec_gpu_context * gpu_ctx;

  for(i=0; i<MAX_OUTPUTS; i++) {
    if(ctx->h_pwrbuf[i]) {
      hipHostFree(ctx->h_pwrbuf[i]);
      ctx->h_pwrbuf[i] = NULL;
    }
    if(ctx->h_icsbuf[i]) {
      hipHostFree(ctx->h_icsbuf[i]);
      ctx->h_icsbuf[i] = NULL;
    }
  }

  if(ctx->gpu_ctx) {
    gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

    if(gpu_ctx->caller_managed) {
      for(i=0; i < ctx->Nb_host; i++) {
        hipHostUnregister(ctx->h_blkbufs[i]);
      }
    } else {
      if(ctx->h_blkbufs) {
        for(i=0; i < ctx->Nb_host; i++) {
          hipHostFree(ctx->h_blkbufs[i]);
        }
        free(ctx->h_blkbufs);
        ctx->h_blkbufs = NULL;
      }
    }

    // Destroy texture object before freeing referenced memory
    hipDestroyTextureObject(gpu_ctx->tex_obj);

    if(gpu_ctx->d_fft_in) {
      hipFree(gpu_ctx->d_fft_in);
    }

    if(gpu_ctx->d_blk_expansion_buf) {
      hipFree(gpu_ctx->d_blk_expansion_buf);
    }
    
    if(gpu_ctx->d_comp4_exp_LUT) {
      hipFree(gpu_ctx->d_comp4_exp_LUT);
      hipDestroyTextureObject(gpu_ctx->comp4_exp_tex_obj);
    }

    if(gpu_ctx->d_work_area) {
      hipFree(gpu_ctx->d_work_area);
    }

    if(gpu_ctx->compute_stream != NO_STREAM) {
      hipStreamDestroy(gpu_ctx->compute_stream);
    }

    if(gpu_ctx->d_fft_out) {
      hipFree(gpu_ctx->d_fft_out);
    }

    for(i=0; i<MAX_OUTPUTS; i++) {
      if(gpu_ctx->d_pwr_out[i]) {
        hipFree(gpu_ctx->d_pwr_out[i]);
      }
      if(gpu_ctx->d_ics_out[i]) {
        hipFree(gpu_ctx->d_ics_out[i]);
      }
      for(p=0; p<2; p++) {
        if(gpu_ctx->plan[i][p] != NO_PLAN) {
          hipfftDestroy(gpu_ctx->plan[i][p]);
        }
      }
    }

    if(ctx->incoherently_sum){
      if(ctx->Aws){
        hipHostFree(ctx->Aws);
      }
      if(gpu_ctx->d_Aws){
        hipFree(gpu_ctx->d_Aws);
      }
    }

    free(ctx->gpu_ctx);
    ctx->gpu_ctx = NULL;
  }
}

// Copy `ctx->h_blkbufs` to GPU input buffer.
// Returns 0 on success, non-zero on error.
int rawspec_copy_blocks_to_gpu_expanding_complex4(rawspec_context * ctx,
  off_t src_idx, off_t dst_idx, size_t num_blocks)
{
  if(num_blocks > ctx->Nb){
    fprintf(stderr, "%s: num_blocks (%lu) > Nb (%u)\n", __FUNCTION__, num_blocks, ctx->Nb);
    return 1;
  }

  int b;
  off_t sblk;
  off_t dblk;
  dim3 grid;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  // Calculated for complex4 samples
  const size_t block_size = (gpu_ctx->guppi_channel_stride * ctx->Nc)/2;

  for(b=0; b < num_blocks; b++) {
    sblk = (src_idx + b) % ctx->Nb_host;
    dblk = (dst_idx + b) % ctx->Nb;
    rc = hipMemcpyAsync(gpu_ctx->d_blk_expansion_buf + (dblk * block_size), ctx->h_blkbufs[sblk],
                          block_size, hipMemcpyHostToDevice, gpu_ctx->compute_stream);

    if(rc != hipSuccess) {
      PRINT_ERRMSG(rc);
      return 1;
    }
  }
  
  // Calculate grid dimensions, fastest to slowest
  const unsigned int thread_count = ctx->Np;

  grid.x = ctx->Ntpb;
  grid.y = ctx->Nc;
  grid.z = num_blocks;
  
  copy_expand_complex4<<<grid, thread_count, 0, gpu_ctx->compute_stream>>>(
                                              gpu_ctx->d_fft_in, gpu_ctx->d_blk_expansion_buf, 
                                              num_blocks, block_size, gpu_ctx->guppi_channel_stride/2);

  return 0;
}

// Copy `ctx->h_blkbufs` to GPU input buffer.
// Returns 0 on success, non-zero on error.
int rawspec_copy_blocks_to_gpu(rawspec_context * ctx,
    off_t src_idx, off_t dst_idx, size_t num_blocks)
{
  int b;
  off_t sblk;
  off_t dblk;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  for(b=0; b < num_blocks; b++) {
    sblk = (src_idx + b) % ctx->Nb_host;
    dblk = (dst_idx + b) % ctx->Nb;

    rc = hipMemcpy2D(gpu_ctx->d_fft_in + dblk * gpu_ctx->guppi_channel_stride,
                      ctx->Nb * gpu_ctx->guppi_channel_stride,  // dpitch
                      ctx->h_blkbufs[sblk],                     // *src
                      gpu_ctx->guppi_channel_stride,            // spitch
                      gpu_ctx->guppi_channel_stride,            // width
                      ctx->Nc,                                  // height
                      hipMemcpyHostToDevice);

    if(rc != hipSuccess) {
      PRINT_ERRMSG(rc);
      return 1;
    }
  }

  return 0;
}

// Sets `num_blocks` blocks to zero in GPU input buffer, starting with block at
// `dst_idx`.  If `dst_idx + num_blocks > cts->Nb`, the zeroed blocks will wrap
// to the beginning of the input buffer, but no processing will occur.  Callers
// should avoid this case as it will likely not give the desired results.
// Returns 0 on success, non-zero on error.
int rawspec_zero_blocks_to_gpu(rawspec_context * ctx,
    off_t dst_idx, size_t num_blocks)
{
  int b;
  off_t dblk;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  for(b=0; b < num_blocks; b++) {
    dblk = (dst_idx + b) % ctx->Nb;

    rc = hipMemset2D(gpu_ctx->d_fft_in + dblk * gpu_ctx->guppi_channel_stride,
                      ctx->Nb * gpu_ctx->guppi_channel_stride,  // pitch
                      0,                                        // value
                      gpu_ctx->guppi_channel_stride,            // width
                      ctx->Nc);                                 // height

    if(rc != hipSuccess) {
      PRINT_ERRMSG(rc);
      return 1;
    }
  }

  return 0;
}

// Launches FFTs of data in input buffer.  Whenever an output product
// integration is complete, the power spectrum is copied to the host power
// output buffer and the user provided callback, if any, is called.  This
// function returns zero on success or non-zero if an error is encountered.
//
// The direction of the FFT is determined by the fft_dir parameter.  If fft_dir
// is less than or equal to zero, an inverse (aka backward) transform is
// performed, otherwise a forward transform is performed.
//
// Processing occurs asynchronously.  Use `rawspec_check_for_completion` to
// see how many output products have completed or
// `rawspec_wait_for_completion` to wait for all output products to be
// complete.  New data should NOT be copied to the GPU until
// `rawspec_check_for_completion` returns `ctx->No` or
// `rawspec_wait_for_completion` returns 0.
int rawspec_start_processing(rawspec_context * ctx, int fft_dir)
{
  int i;
  int p;
  int d;
  float * dst;
  size_t dpitch;
  float * src;
  size_t c;
  size_t spitch;
  size_t width;
  size_t height;
  hipfftHandle plan;
  hipError_t cuda_rc;
  hipfftResult cufft_rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;
  size_t fft_outbuf_length;
  dim3 grid_ics;
  const size_t Nchan_per_antenna = ctx->Nc/ctx->Nant;
  const size_t Nantenna_per_chunk = ctx->Ncc/Nchan_per_antenna;

  // Increment inbuf_count
  gpu_ctx->inbuf_count++;

  // For each output product
  for(i=0; i < ctx->No; i++) {
    // Length of an FFT output buffer when abs(Npotout)==4, must be 0 when
    // Npolout==1
    fft_outbuf_length = ctx->Npolout[i] == 1 ? 0 : ctx->Nb*ctx->Ntpb*ctx->Ncc;
    for(c=0; c < ctx->Nc; c += ctx->Ncc) {

      // For each input polarization
      for(p=0; p < ctx->Np; p++) {
        // Get plan
        plan = gpu_ctx->plan[i][p];

        // Add FFT to stream
        cufft_rc = hipfftExecC2C(plan,
                                ((hipfftComplex *)gpu_ctx->d_fft_in) + p + (c * ctx->Nb * ctx->Ntpb * ctx->Np),
                                gpu_ctx->d_fft_out + p * fft_outbuf_length,
                                fft_dir <= 0 ? HIPFFT_BACKWARD : HIPFFT_FORWARD);

        if(cufft_rc != HIPFFT_SUCCESS) {
          PRINT_ERRMSG(cufft_rc);
          return 1;
        }
      }

      // If time to dump
      if(gpu_ctx->inbuf_count % gpu_ctx->Nis[i] == 0) {
        // If the number of spectra to dump per input buffer is less than the
        // number of spectra per input buffer, then we need to accumulate the
        // sub-integrations together.
        if(ctx->Nds[i] < gpu_ctx->Nss[i]) {
          for(p=0; p < abs(ctx->Npolout[i]); p++) {
            accumulate<<<gpu_ctx->grid[i],
                        gpu_ctx->nthreads[i],
                        0, gpu_ctx->compute_stream>>>
                          (
                            gpu_ctx->d_pwr_out[i] + p*ctx->Nb*ctx->Ntpb*ctx->Ncc,
                            MIN(ctx->Nas[i], gpu_ctx->Nss[i]), // Na
                            ctx->Nts[i],                       // xpitch
                            ctx->Nas[i]*ctx->Nts[i],           // ypitch
                            ctx->Nb*ctx->Ntpb                  // zpitch
                          );
          }
        }

        if(ctx->incoherently_sum){
          grid_ics.x = (ctx->Nts[i] * ctx->Ncc)/gpu_ctx->nthreads[i];
          grid_ics.y = abs(ctx->Npolout[i]);
          grid_ics.z = ctx->Nds[i];
          
          incoherent_sum<<<grid_ics, gpu_ctx->nthreads[i], 0, gpu_ctx->compute_stream>>>(
                                        gpu_ctx->d_pwr_out[i], gpu_ctx->d_ics_out[i] + (c%Nchan_per_antenna)*ctx->Nts[i], gpu_ctx->d_Aws + c/Nchan_per_antenna,
                                        Nantenna_per_chunk, ctx->Nts[i],
                                        ctx->Nb*ctx->Ntpb*ctx->Nc/ctx->Nant, // Antenna pitch
                                        ctx->Nb*ctx->Ntpb, // Coarse Channel pitch
                                        ctx->Nb*ctx->Ntpb*ctx->Ncc, // Polarisation pitch
                                        ctx->Nts[i]*ctx->Nas[i], // Spectra pitch
                                        
                                        ctx->Nts[i], // Coarse Channel pitch for ics
                                        ctx->Nts[i]*ctx->Nc/ctx->Nant, // Polarisation pitch for ics
                                        abs(ctx->Npolout[i]) * ctx->Nts[i] * ctx->Nc/ctx->Nant // Spectra pitch for ics
                                        );
        
          if(c + ctx->Ncc >= ctx->Nc){
            // Copy store_cb_data_t array from host to device
            cuda_rc = hipMemcpyAsync(ctx->h_icsbuf[i],
              gpu_ctx->d_ics_out[i],
              ctx->h_pwrbuf_size[i]/ctx->Nant,
              hipMemcpyDeviceToHost,
              gpu_ctx->compute_stream);
            if(cuda_rc != hipSuccess) {
              PRINT_ERRMSG(cuda_rc);
              return 1;
            }
          }
        }

        if(c + ctx->Ncc >= ctx->Nc){
          // Add pre-dump stream callback
          cuda_rc = hipStreamAddCallback(gpu_ctx->compute_stream,
                                          pre_dump_stream_callback,
                                          (void *)&gpu_ctx->dump_cb_data[i], 0);

          if(cuda_rc != hipSuccess) {
            PRINT_ERRMSG(cuda_rc);
            return 1;
          }
        }

        for(p=0; p < abs(ctx->Npolout[i]); p++) {
          // Copy integrated power spectra (or spectrum) to host.  This is done as
          // two 2D copies to get channel 0 in the center of the spectrum.  Special
          // care is taken in the unlikely event that Nt is odd.
          src    = gpu_ctx->d_pwr_out[i] + p*ctx->Nb*ctx->Ntpb*ctx->Ncc;
          dst    = ctx->h_pwrbuf[i] + (p*ctx->Nts[i]*ctx->Nc) + (c*ctx->Nts[i]);
          spitch = gpu_ctx->Nss[i] * ctx->Nts[i] * sizeof(float);
          dpitch = ctx->Nts[i] * sizeof(float);
          height = ctx->Ncc;

          for(d=0; d < ctx->Nds[i]; d++) {

            // Lo to hi
            width  = ((ctx->Nts[i]+1) / 2) * sizeof(float);
            cuda_rc = hipMemcpy2DAsync(dst + ctx->Nts[i]/2,
                                        dpitch,
                                        src,
                                        spitch,
                                        width,
                                        height,
                                        hipMemcpyDeviceToHost,
                                        gpu_ctx->compute_stream);

            if(cuda_rc != hipSuccess) {
              PRINT_ERRMSG(cuda_rc);
              rawspec_cleanup(ctx);
              return 1;
            }

            // Hi to lo
            width  = (ctx->Nts[i] / 2) * sizeof(float);
            cuda_rc = hipMemcpy2DAsync(dst,
                                        dpitch,
                                        src + (ctx->Nts[i]+1) / 2,
                                        spitch,
                                        width,
                                        height,
                                        hipMemcpyDeviceToHost,
                                        gpu_ctx->compute_stream);

            if(cuda_rc != hipSuccess) {
              PRINT_ERRMSG(cuda_rc);
              rawspec_cleanup(ctx);
              return 1;
            }

            // Increment src and dst pointers
            src += ctx->Nts[i] * ctx->Nas[i];
            dst += abs(ctx->Npolout[i]) * ctx->Nts[i] * ctx->Nc;
          }
        }

        if(c + ctx->Ncc >= ctx->Nc){
          // Add post-dump stream callback
          cuda_rc = hipStreamAddCallback(gpu_ctx->compute_stream,
                                          post_dump_stream_callback,
                                          (void *)&gpu_ctx->dump_cb_data[i], 0);

          if(cuda_rc != hipSuccess) {
            PRINT_ERRMSG(cuda_rc);
            return 1;
          }
        }

        // Add power buffer clearing hipMemset call to stream
        cuda_rc = hipMemsetAsync(gpu_ctx->d_pwr_out[i], 0,
                                  abs(ctx->Npolout[i])*ctx->Nb*ctx->Ntpb*ctx->Ncc*sizeof(float),
                                  gpu_ctx->compute_stream);

        if(cuda_rc != hipSuccess) {
          PRINT_ERRMSG(cuda_rc);
          return 1;
        }
        if(c + ctx->Ncc >= ctx->Nc && ctx->incoherently_sum){
          // Add ics buffer clearing hipMemset call to stream
          cuda_rc = hipMemsetAsync(gpu_ctx->d_ics_out[i], 0,
                                    abs(ctx->Npolout[i])*ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float)/ctx->Nant,
                                    gpu_ctx->compute_stream);
    
          if(cuda_rc != hipSuccess) {
            PRINT_ERRMSG(cuda_rc);
            return 1;
          }
        }

      } // If time to dump
    } // For each chunk of channels
  } // For each output product

  return 0;
}

int rawspec_copy_blocks_to_gpu_and_start_processing(rawspec_context * ctx, size_t num_blocks, char expand4bps_to8bps, int fft_dir)
{
  if(expand4bps_to8bps){
    rawspec_copy_blocks_to_gpu_expanding_complex4(ctx, 0, 0, num_blocks);
  }
  else{
    rawspec_copy_blocks_to_gpu(ctx, 0, 0, num_blocks);
  }
  return rawspec_start_processing(ctx, fft_dir);
}

// Waits for any processing to finish, then clears output power buffers and
// resets inbuf_count to 0.  Returns 0 on success, non-zero on error.
int rawspec_reset_integration(rawspec_context * ctx)
{
  int i;
  hipError_t cuda_rc;
  rawspec_gpu_context * gpu_ctx;

  // Mae sure gpu_ctx exists
  if(!ctx->gpu_ctx) {
    return 1;
  }
  gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  // Wait for any/all pending work to complete
  rawspec_wait_for_completion(ctx);

  // For each output product
  for(i=0; i < ctx->No; i++) {
    // Clear power output buffer
    cuda_rc = hipMemset(gpu_ctx->d_pwr_out[i], 0,
        abs(ctx->Npolout[i])*ctx->Nb*ctx->Ntpb*ctx->Nc*sizeof(float));
    if(cuda_rc != hipSuccess) {
      PRINT_ERRMSG(cuda_rc);
      return 0;
    }
  }

  // Reset inbuf_count
  gpu_ctx->inbuf_count = 0;

  return 0;
}

// Returns true if the "compute stream" is done processing.
unsigned int rawspec_check_for_completion(rawspec_context * ctx)
{
  int complete = 0;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  rc = hipStreamQuery(gpu_ctx->compute_stream);
  if(rc == hipSuccess) {
    complete++;
  }

  return complete;
}

// Waits for any pending output products to be compete processing the current
// input buffer.  Returns zero when complete, non-zero on error.
int rawspec_wait_for_completion(rawspec_context * ctx)
{
  int i = 0;
  hipError_t rc;
  rawspec_gpu_context * gpu_ctx = (rawspec_gpu_context *)ctx->gpu_ctx;

  for(i=0; i < ctx->No; i++) {
    // Add one final pre-dump stream callback to ensure final output thread can
    // be joined.
    rc = hipStreamAddCallback(gpu_ctx->compute_stream, pre_dump_stream_callback,
                                    (void *)&gpu_ctx->dump_cb_data[i], 0);
    if(rc != hipSuccess) {
      PRINT_ERRMSG(rc);
      return 1;
    }
  }

  rc = hipStreamSynchronize(gpu_ctx->compute_stream);
  if(rc != hipSuccess) {
    return 1;
  }

  return 0;
}
